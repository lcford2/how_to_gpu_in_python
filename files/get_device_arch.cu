#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Display a variety of information on the first CUDA device in this system,
 * including driver version, runtime version, compute capability, bytes of
 * global memory, etc.
 */

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
  }

int main(int argc, char **argv) {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("There are no devices that support CUDA\n");
        exit(EXIT_SUCCESS);
    }
    
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);

    int dev = 0;
    CHECK(hipSetDevice(dev));
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    printf("Please use: sm_%d%d\n", deviceProp.major, deviceProp.minor);

    exit(EXIT_SUCCESS);
}
